#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include "shared.hpp"

/* Get the position of a two dimensional flat array. */
#define ARRAY2_IDX(a,b,size) (a * size) + b

/* Get the index via the CUDA block and thread index. */
#define IDX(bIdx,bDim,tIdx,size) ARRAY2_IDX((bIdx.y * bDim.y + tIdx.y),((bIdx.x * bDim.x) + tIdx.x), size)

/* Return if the idx is out of bounds of the array size. */ 
#define IDX_GUARD(size_x, size_y, idx) if((size_x * size_y) <= idx) return

/* Basic inlined math operations for the rgb format. */
#define MAXRGB8(r,g,b) ((uint8_t) fmaxf(fmaxf((float)r, (float)g), (float)b))
#define MINRGB8(r,g,b) ((uint8_t) fminf(fminf((float)r, (float)g), (float)b))

/*
 * CUDA kernel for the grayscaling operation.
 */
__global__ void op_kernel_grey(uint32_t width, uint32_t height, uint32_t *in, uint32_t *out)
{
	uint32_t idx = IDX(blockIdx, blockDim, threadIdx, width);

	IDX_GUARD(width, height, idx);

	uint8_t color = 
		  (0.21 * RED8(in[idx]))
		+ (0.72 * GREEN8(in[idx]))
		+ (0.07 * BLUE8(in[idx]));

	uint8_t alpha = ALPHA8(in[idx]);

	out[idx] = RGBA32(color, color, color, alpha);
}
#define OP_KERNEL_GREY 1

/*
 * CUDA kernel for converting the rgba to hsv colorspace.
 */
 __global__ void op_kernel_hsv(uint32_t width, uint32_t height, uint32_t *in, uint32_t *out)
{
	uint32_t idx = IDX(blockIdx, blockDim, threadIdx, width);

	IDX_GUARD(width, height, idx);

	int32_t next = idx * 3; // hsv has only 3 channels
	
	/* Normalize color values except alpha. */
	uint8_t red = RED8(in[idx]);
	uint8_t green = GREEN8(in[idx]);
	uint8_t blue = BLUE8(in[idx]);

	/* Calulate conversion parameters */
	uint8_t cmax = MAXRGB8(red, green, blue);
	uint8_t cmin = MINRGB8(red, green, blue);
	uint8_t diff = cmax - cmin;

	/* Calculate hue. */
	uint8_t hue = 0;

	if(diff != 0)
	{
		if(cmax == red)
			hue = 43 * ((green - blue) / diff);
		else if(cmax == green)
			hue = 85 + 43 * ((blue - red) / diff);
		else
			hue = 171 + 43 * ((red - green) / diff);
	}

	/* Calculate saturation. */
	uint8_t saturation = 0;

	if(cmax != 0)
		saturation = 255 * diff / cmax;
	
	/* Calculate value. */
	uint8_t value = cmax;

	/* Write only three channels. */
	((uint8_t *)out)[next++] = hue;
	((uint8_t *)out)[next++] = saturation;
	((uint8_t *)out)[next++] = value;
}
#define OP_KERNEL_HSV 2

/*
 * CUDA kernel for the gaussian blur operation.
 */
 __global__ void op_kernel_blur(uint32_t width, uint32_t height, uint32_t *in, uint32_t *out)
{
	uint32_t x = (blockIdx.x * blockDim.x) + threadIdx.x;
	uint32_t y = (blockIdx.y * blockDim.y) + threadIdx.y;

	uint32_t idx = y * width + x;

	IDX_GUARD(width, height, idx);

	const int32_t filter_size = 5;
	const int32_t filter_pivot = filter_size / 2;

	float filter[filter_size][filter_size] =
	{
		{1.0f,  4.0f,  6.0f,  4.0f,  1.0f},
		{4.0f, 16.0f, 24.0f, 16.0f,  4.0f},
		{6.0f, 24.0f, 36.0f, 24.0f,  6.0f},
		{4.0f, 16.0f, 24.0f, 16.0f,  4.0f},
		{1.0f,  4.0f,  6.0f,  4.0f,  1.0f}
	};

	float filter_factor = 1.0f / 256.0f;
	float filter_bias = 0.0f;

	float red = 0, green = 0, blue = 0, alpha = 0;

	for(int32_t filter_y = 0; filter_y < filter_size; ++filter_y)
	{
		int32_t filter_y_idx = x - filter_pivot + filter_y;

		if(filter_y_idx < 0 || filter_y_idx >= height)
			continue;

		for(int32_t filter_x = 0; filter_x < filter_size; ++filter_x)
		{
			int32_t filter_x_idx = y - filter_pivot + filter_x;

			if(filter_x_idx < 0 || filter_x_idx >= width)
				continue;

			int32_t filter_idx = ARRAY2_IDX(filter_y_idx, filter_x_idx, width);

			red += filter[filter_y][filter_x] * ((float)RED8(in[filter_idx]));
			green += filter[filter_y][filter_x] * ((float)GREEN8(in[filter_idx]));
			blue += filter[filter_y][filter_x] * ((float)BLUE8(in[filter_idx]));
			alpha += filter[filter_y][filter_x] * ((float)ALPHA8(in[filter_idx]));
		}
	}

	red = TRUNCATE_CHANNEL(red, filter_factor, filter_bias);
	green = TRUNCATE_CHANNEL(green, filter_factor, filter_bias);
	blue = TRUNCATE_CHANNEL(blue, filter_factor, filter_bias);
	alpha = TRUNCATE_CHANNEL(alpha, filter_factor, filter_bias);

	out[idx] = RGBA32((uint8_t)red, (uint8_t)green, (uint8_t)blue, (uint8_t)alpha);
}
#define OP_KERNEL_BLUR 3


/*
 * Get the number of CUDA blocks. 
 */
dim3 cuda_blocks(uint32_t x, uint32_t y, dim3 threads)
{
	dim3 blocks((x / threads.x + 1), (y / threads.y + 1));
	return blocks;
}

/*
 * Print CUDA error and return failure status code.
 */
#define CUDA_ERROR_CHECK(call) {\
	hipError_t error = call;\
	if(error != hipSuccess)\
	{\
		printf("CUDA error %d in %s line %d: %s", error, __FILE__, __LINE__, hipGetErrorString(error));\
		return EXIT_FAILURE;\
	}\
}\

/*
 * Executes and distributes the specified kernel.
 */
int execute_cuda_kernel(uint32_t kernel, uint32_t width, uint32_t height, uint32_t *data)
{
	size_t size = sizeof(uint32_t) * width * height;

	uint32_t *in, *out; /* avoid undefined behaviour, see http://www.c-faq.com/ptrs/genericpp.html */

	/* Allocate CUDA buffers. */
	CUDA_ERROR_CHECK(hipMalloc((void **) &in, size));
	CUDA_ERROR_CHECK(hipMalloc((void **) &out, size));
	CUDA_ERROR_CHECK(hipMemcpy(in, data, size, hipMemcpyHostToDevice));

	/* Define distribution levels. */
	dim3 threads(32,32);
	dim3 blocks = cuda_blocks(width, height, threads);

	/* Execute the specified CUDA kernel. */
	switch(kernel)
	{
		case OP_KERNEL_GREY:
			op_kernel_grey<<<threads, blocks>>>(width, height, in, out);
			break;
		case OP_KERNEL_BLUR:
			op_kernel_blur<<<threads, blocks>>>(width, height, in, out);
			break;
		case OP_KERNEL_HSV:
			op_kernel_hsv<<<threads, blocks>>>(width, height, in, out);
			break;
		default:
			return EXIT_FAILURE;
	}

	/* Copy CUDA buffer back to source array and free allocated buffers. */
	CUDA_ERROR_CHECK(hipMemcpy(data, out, size, hipMemcpyDeviceToHost));
	CUDA_ERROR_CHECK(hipFree(in));
	CUDA_ERROR_CHECK(hipFree(out));

	return EXIT_SUCCESS;
}

/*
 * Greyscales the colors of the image.
 */
int op_grey(uint32_t width, uint32_t height, uint32_t *data)
{
	return execute_cuda_kernel(OP_KERNEL_GREY, width, height, data);
}

/*
 * Converts the colorspace from rgba to hsv.
 */
int op_hsv(uint32_t width, uint32_t height, uint32_t *data)
{
	return execute_cuda_kernel(OP_KERNEL_HSV, width, height, data);
}

/*
 * Applies a emboss filter to the image.
 */
int op_emboss(uint32_t width, uint32_t height, uint32_t *data)
{
	return execute_cuda_kernel(0, width, height, data);
}

/*
 * Applies a gaussian blur filter to the image.
 */
int op_blur(uint32_t width, uint32_t height, uint32_t *data)
{
	return execute_cuda_kernel(OP_KERNEL_BLUR, width, height, data);
}