#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "shared.hpp"

/* Get the position of a two dimensional flat array. */
#define ARRAY2_IDX(a,b,size) (a * size) + b

/* Get the index via the CUDA block and thread index. */
#define IDX(bIdx,bDim,tIdx,size) ARRAY2_IDX((bIdx.y * bDim.y + tIdx.y),((bIdx.x * bDim.x) + tIdx.x), size)

/* Return if the idx is out of bounds of the array size. */ 
#define IDX_GUARD(size_x, size_y, idx) if((size_x * size_y) <= idx) return


/*
 * Get the number of CUDA threads. 
 */
dim3 cuda_threads()
{
	dim3 threads(32,32);
	return threads;
}

/*
 * Get the number of CUDA blocks. 
 */
dim3 cuda_blocks(uint32_t x, uint32_t y, dim3 threads)
{
	dim3 blocks((x / threads.x + 1), (y / threads.y + 1));
	return blocks;
}

/*
 * Allocate generic CUDA memory.
 */
uint32_t* allocate_cuda_buffer(uint32_t x, uint32_t y)
{
	size_t size = sizeof(uint32_t) * x * y;
	uint32_t *buffer;
	hipMalloc((void **) &buffer, size);
	return buffer;
}

/*
 * Allocate generic CUDA memory and copy the given data to the memory.
 */
uint32_t* allocate_cuda__input_buffer(uint32_t x, uint32_t y, uint32_t *data)
{
	size_t size = sizeof(uint32_t) * x * y;
	uint32_t *buffer = allocate_cuda_buffer(x, y);
	hipMemcpy(buffer, data, size, hipMemcpyHostToDevice);
	return buffer;
}

/*
 * Greyscales the colors of the image.
 */
int op_grey(uint32_t width, uint32_t height, uint32_t *data)
{
	dim3 threads = cuda_threads();
	dim3 blocks = cuda_blocks(width, height, threads);
	uint32_t *in = allocate_cuda__input_buffer(width, height, data);
	uint32_t *out = allocate_cuda_buffer(width, height);

	op_kernel_grey<<<threads, blocks>>>(width, height, in, out);
	return EXIT_SUCCESS;
}

/*
 * CUDA kernel for the grayscaling operation.
 */
__global__ void op_kernel_grey(uint32_t width, uint32_t height, uint32_t *in, uint32_t *out)
{
	uint32_t idx = IDX(blockIdx, blockDim, threadIdx, width);

	IDX_GUARD(width, height, idx);

	uint8_t color = 
		  (0.21 * RED8(in[idx]))
		+ (0.72 * GREEN8(in[idx]))
		+ (0.07 * BLUE8(in[idx]));

	uint8_t alpha = ALPHA8(in[idx]);

	out[idx] = RGBA32(color, color, color, alpha);
}


/*
 * Required filter information.
 */
#define filter_size 5
#define filter_pivot filter_size / 2
#define filter_factor = 1.0f / 256.0f;
#define filter_bias = 0.0f;
__device__ float filter[filter_size][filter_size];

/*
 * Applies a gaussian blur filter to the image.
 */
int op_blur(uint32_t width, uint32_t height, uint32_t *data)
{
	float filter_data[filter_size][filter_size] = 
	{
		{1.0f,  4.0f,  6.0f,  4.0f,  1.0f},
		{4.0f, 16.0f, 24.0f, 16.0f,  4.0f},
		{6.0f, 24.0f, 36.0f, 24.0f,  6.0f},
		{4.0f, 16.0f, 24.0f, 16.0f,  4.0f},
		{1.0f,  4.0f,  6.0f,  4.0f,  1.0f}
	};

	hipMemcpyToSymbol(HIP_SYMBOL(filter), filter_data, filter_size * filter_size * sizeof(float));

	dim3 threads = cuda_threads();
	dim3 blocks = cuda_blocks(width, height, threads);
	uint32_t *in = allocate_cuda__input_buffer(width, height, data);
	uint32_t *out = allocate_cuda_buffer(width, height);

	op_kernel_blur<<<threads, blocks>>>(width, height, in, out);

	return EXIT_SUCCESS;
}

#define TRUNCATE_CHANNEL(value,factor,bias) std::min(std::max(factor * value + bias, 0.0f), 255.0f)

/*
 * CUDA kernel for the gaussian blur operation.
 */
 __global__ void op_kernel_blur(uint32_t width, uint32_t height, uint32_t *in, uint32_t *out)
{
	uint32_t x = (blockIdx.x * blockDim.x) + threadIdx.x;
	uint32_t y = (blockIdx.y * blockDim.y) + threadIdx.y;

	uint32_t idx = y * width + x;

	IDX_GUARD(width, height, idx);

	float red = 0, green = 0, blue = 0, alpha = 0;

	for(int32_t filter_y = 0; filter_y < filter_size; ++filter_y)
	{
		int32_t filter_y_idx = x - filter_pivot + filter_y;

		if(filter_y_idx < 0 || filter_y_idx >= height)
			continue;

		for(int32_t filter_x = 0; filter_x < filter_size; ++filter_x)
		{
			int32_t filter_x_idx = y - filter_pivot + filter_x;

			if(filter_x_idx < 0 || filter_x_idx >= width)
				continue;

			int32_t filter_idx = ARRAY2_IDX(filter_y_idx, filter_x_idx, width);

			red += filter[filter_y][filter_x] * ((float)RED8(in[filter_idx]));
			green += filter[filter_y][filter_x] * ((float)GREEN8(in[filter_idx]));
			blue += filter[filter_y][filter_x] * ((float)BLUE8(in[filter_idx]));
			alpha += filter[filter_y][filter_x] * ((float)ALPHA8(in[filter_idx]));
		}
	}

	red = TRUNCATE_CHANNEL(red, filter_factor, filter_bias);
	green = TRUNCATE_CHANNEL(green, filter_factor, filter_bias);
	blue = TRUNCATE_CHANNEL(blue, filter_factor, filter_bias);
	alpha = TRUNCATE_CHANNEL(alpha, filter_factor, filter_bias);

	out[index] = RGBA32((uint8_t)red, (uint8_t)green, (uint8_t)blue, (uint8_t)alpha);
}

int op_hsv(uint32_t width, uint32_t height, uint32_t *data)
{
	return EXIT_SUCCESS;
}

int op_emboss(uint32_t width, uint32_t height, uint32_t *data)
{
	return EXIT_SUCCESS;
}

